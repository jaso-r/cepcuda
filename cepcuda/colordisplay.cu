#include "hip/hip_runtime.h"
#ifndef __ceptools_colordisplay_cu
#define __ceptools_colordisplay_cu

/*
 *  colordisplay.cu
 *
 *  CUDA code used to modify the texture image displaying algorithm results.
 */

#include "hip/hip_runtime.h"
#include ""

#include <stdlib.h>
#include <stdio.h>
#include "cepdb.h"
#include "globals.h"

#define THREAD_CNT 512

texture<float, 2> elevTex;
hipArray *d_elev_array;

// This will output the proper CUDA error strings in the event that a CUDA host call returns an error
#define checkCudaErrors(err)           __checkCudaErrors (err, __FILE__, __LINE__)

inline void __checkCudaErrors(hipError_t err, const char *file, const int line)
{
	if (hipSuccess != err)
	{
		fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",
			file, line, (int)err, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

// kernel sets the "color" value based off of caloric cost if it has been calculated or elevation otherwise.
__global__ void color_kernel(float *od, float *d_cals, int len, int width, float minElevation, float maxElevation, float bandSize)
{
	int arrayIndex = (blockIdx.x * blockDim.x) + threadIdx.x;
	int xIndex = arrayIndex % width;
	int yIndex = arrayIndex / width;

	if (arrayIndex < len)
	{
		float cal = d_cals[arrayIndex];
		if (cal >= 0)
		{
			od[arrayIndex] = ((float)(((int)cal) % ((int)bandSize))) / bandSize;
		}
		else
		{
			od[arrayIndex] = (tex2D(elevTex, xIndex, yIndex) - minElevation) / (maxElevation - minElevation);
		}
	}
}

// called externally to initialize the textures used by the kernel
extern "C"
void initTexture(int width, int height, void *elevVals)
{
	int size = width * height * sizeof(float);

	// copy image data to array
	hipChannelFormatDesc channelDesc;
	channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	checkCudaErrors(hipMallocArray(&d_elev_array, &channelDesc, width, height));
	checkCudaErrors(hipMemcpyToArray(d_elev_array, 0, 0, elevVals, size, hipMemcpyHostToDevice));

	// set texture parameters
	elevTex.addressMode[0] = hipAddressModeClamp;
	elevTex.addressMode[1] = hipAddressModeClamp;
	elevTex.filterMode = hipFilterModePoint;
	elevTex.normalized = false;

	// Bind the array to the texture
	checkCudaErrors(hipBindTextureToArray(elevTex, d_elev_array, channelDesc));
}

// called externally to cleanup the textures used by the kernel
extern "C"
void freeTextures()
{
	checkCudaErrors(hipFreeArray(d_elev_array));
}

// called externally to run the kernel and update the texture image based off of the currently calculated calorie data
// and the elevation data.
extern "C"
void convertToColor(float *d_dest, float *h_cals, int width, int height, float minElevation, float maxElevation, float bandSize)
{
	// sync host and start computation timer_kernel
	checkCudaErrors(hipDeviceSynchronize());
	checkCudaErrors(hipBindTextureToArray(elevTex, d_elev_array));

	float *d_cals = NULL;
	checkCudaErrors(hipMalloc((void **)&d_cals, width * height * sizeof(float)));
	checkCudaErrors(hipMemcpy(d_cals, h_cals, width * height * sizeof(float), hipMemcpyHostToDevice));

	int blockCount = (width * height) / THREAD_CNT;
	if ((width * height) % THREAD_CNT != 0) blockCount++;

	color_kernel<<<blockCount, THREAD_CNT>>>(d_dest, d_cals, (width * height), width, minElevation, maxElevation, bandSize);
	checkCudaErrors(hipDeviceSynchronize());

	checkCudaErrors(hipFree(d_cals));
}


#endif